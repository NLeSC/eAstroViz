#include <stdio.h>
#include <time.h>
#include <limits.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <string.h>
#include <stdlib.h>
#include <arpa/inet.h>
#include <float.h>
#include "Data_reader.h"
#include "Device_data.h"
#include "Data_info.h"


#define MAX_SUBBANDS INT_MAX
#define MAX_SEQ_NR   INT_MAX
#define TRUE         1
#define FALSE        0


/* Converts four big endian ordered bytes into a float */
float bytes2floatBE(unsigned char *raw){
  int i;
  union{
    unsigned char bytes[4];
    float fp;
  } un ;
  for(i=0;i<4;i++){
    un.bytes[i] = raw[3-i];
  }
  return un.fp;
}

/* Converts four little endian ordered bytes into a float */
float bytes2floatLE(unsigned char *raw){
  int i;
  union{
    unsigned char bytes[4];
    float fp;
  } un ;
  for(i=0;i<4;i++){
    un.bytes[i] = raw[i];
  }
  return un.fp;
}

void remove_initial_flagged(Data_info* h_data_info){
  long initial_flagged_count = 0;
  int time;
  int subband;
  int channel;
  int pol;

  for(time = 0;time < h_data_info->nr_times; time++){
    for(subband = 0; subband < h_data_info->nr_subbands; subband++){
      for(channel = 0; channel < h_data_info->nr_channels; channel++){
        for(pol = 0; pol < h_data_info->nr_polarizations; pol++){
          if(h_data_info->initial_flagged[time][subband][channel]){
            h_data_info->data[time][subband][pol][channel] = 0.0f;
            initial_flagged_count++;
          } 
        }
      }
    }
  }
}



/* Allocates memory */
void allocate_memory(float***** data, unsigned char**** flagged, 
    unsigned char**** initial_flagged, int nr_times, int nr_subbands, 
    int nr_channels, int nr_polarizations){
  
  int i,j,k;
  *data = (float****) malloc(sizeof(float***) * nr_times);
  if(*data == NULL){
    perror("Error allocating memory for data buffer:");
    exit(1);
  }
  *flagged = (unsigned char***) malloc(sizeof(unsigned char **) * nr_times);
  if(*flagged == NULL){

    perror("Error allocating memory for flag buffer:");
    exit(1);
  }
  *initial_flagged = (unsigned char***) malloc(sizeof(unsigned char **) * nr_times);
  if(*initial_flagged == NULL){
    perror("Error allocating memory for flag buffer:");
    exit(1);
  }

  for(i=0;i<nr_times;i++){
    (*data)[i] = (float ***) malloc(sizeof(float**) * nr_subbands);
    if((*data)[i] == NULL){
      perror("Error allocating memory for data on time");
      exit(1);
    }
    (*flagged)[i] = (unsigned char **) malloc(sizeof(unsigned char *) * nr_subbands);
    if((*flagged)[i] == NULL){
      perror("Error allocating memory for flags on time");
      exit(1);
    }
    (*initial_flagged)[i] = (unsigned char **) malloc(sizeof(unsigned char *) 
    * nr_subbands);
    if((*initial_flagged)[i] == NULL){
      perror("Error allocating memory for initial_flags on time");
      exit(1);
    }

    for(j=0;j<nr_subbands;j++){
      (*data)[i][j] = (float **) malloc(sizeof(float*) * nr_polarizations);
      if((*data)[i][j] == NULL){
        perror("Error allocating memory for data on subband at time");
        exit(1);
      }

      for(k=0;k<nr_polarizations;k++){
        (*data)[i][j][k] = (float *) malloc(sizeof(float) * nr_channels);
        if((*data)[i][j][k] == NULL){
          perror("Error allocating memory for data on on polarization on subband at time ");
          exit(1);
        }
        memset((*data)[i][j][k], 0, sizeof(float) * nr_channels);
      }
      (*flagged)[i][j] = (unsigned char*) malloc(sizeof(unsigned char) * nr_channels);
      if((*flagged)[i][j] == NULL){
        perror("Error allocating memory for flags on subband at time");
        exit(1);
      }
      (*initial_flagged)[i][j] = (unsigned char*) malloc(sizeof(unsigned char) *
      nr_channels);
      if((*initial_flagged)[i][j] == NULL){
        perror("Error allocating memory for initial flags on subband at time");
        exit(1);
      }
      memset((*flagged)[i][j], 0, sizeof(unsigned char) * nr_channels);
      memset((*initial_flagged)[i][j], 0, sizeof(unsigned char) * nr_channels);
    }
  }
}

Data_info* read_file(char* file_name, int* meta_data, int little_endian)
{
  int nr_stations;
  int nr_times;
  int nr_subbands;
  int nr_subbands_in_file;
  int nr_channels;
  int nr_polarizations;
  int integration_factor = 1;
  int station_block_size;
  FILE* fin;
  int second;
  int time;
  int subband;
  int channel;
  int pol;
  unsigned char* byte_buffer;
  float sample;
  int index;
  int err;
  float**** data;
  unsigned char*** flagged;
  unsigned char*** initial_flagged;
  Data_info* data_info;


  /*open the file*/

  fin = fopen(file_name, "r");
  if(fin == NULL){
    perror("Error opening file:");
    exit(1);
  }

  if(meta_data == NULL){

    /* Read the number of stations */
    if((fread(&nr_stations, sizeof(nr_stations), 1, fin)) == 0){
      perror("Error reading number of stations:");
      exit(1);
    }
    /* Convert to little endianness, as file is stored big endian */
    nr_stations = htonl(nr_stations);
    
    /* Read the number of times*/
    if((fread(&nr_times, sizeof(nr_times), 1, fin)) == 0){
      perror("Error reading number of times:");
      exit(1);
    }
    /* Convert to little endianness, as file is stored big endian */
    nr_times = htonl(nr_times);
    nr_times /= integration_factor;

    /* Read the number of subbands */
    if((fread(&nr_subbands_in_file, sizeof(nr_subbands_in_file), 1, fin)) == 0){
      perror("Error reading number of subbands:");
      exit(1);
    }
    /* Convert to little endianness, as file is stored big endian */
    nr_subbands_in_file = htonl(nr_subbands_in_file);

    /* Read the number of channels */
    if((fread(&nr_channels, sizeof(nr_channels), 1, fin)) == 0){
      perror("Error reading number of channels:");
      exit(1);
    }
    /* Convert to little endianness, as file is stored big endian */
    nr_channels = htonl(nr_channels);

    if((fread(&nr_polarizations, sizeof(nr_polarizations), 1, fin)) == 0){
      perror("Error reading number of polarizations:");
      exit(1);
    }
    /* Convert to little endianness, as file is stored big endian */
    nr_polarizations = htonl(nr_polarizations);

  }else{

    nr_stations         = meta_data[0];
    nr_times            = meta_data[1];
    nr_subbands_in_file = meta_data[2];
    nr_channels         = meta_data[3];
    nr_polarizations    = meta_data[4];

  }


  //printf("Parameters read! nr_stations: %d, nr_times: %d, nr_subbands: %d, nr_channels: %d, nr_polarizations: %d\n", nr_stations, nr_times, nr_subbands_in_file, nr_channels, nr_polarizations);

  nr_subbands = nr_subbands_in_file;
  if(MAX_SUBBANDS < nr_subbands_in_file){
    nr_subbands = MAX_SUBBANDS;
  }

  
  /* allocate memory for data and flags. */
  allocate_memory(&data, &flagged, &initial_flagged, nr_times, nr_subbands, nr_channels, nr_polarizations);

  station_block_size = integration_factor * nr_subbands_in_file * nr_channels *
                       nr_polarizations * sizeof(float);


  byte_buffer =  (unsigned char*) malloc(sizeof(unsigned char) * station_block_size);
  if(byte_buffer == NULL){
    perror("Error allocating memory for byte_buffer:");
    exit(1);
  }


  for(second = 0; second < nr_times; second ++){
    if(second > MAX_SEQ_NR){
      break;
    }
    err = fread(byte_buffer, sizeof(unsigned char), station_block_size, fin);
    if(err != station_block_size){
      perror("Error reading bytes into buffer");
      exit(1);
    }
        
    index = 0;
    
    for(time = 0; time < integration_factor; time++){
      for(subband = 0; subband < nr_subbands_in_file; subband++){
        for(channel = 0; channel < nr_channels; channel++){
          for(pol = 0; pol < nr_polarizations; pol++){
            if(little_endian){
              sample = bytes2floatLE(&byte_buffer[index * sizeof(float)]);;
            }else{
              sample = bytes2floatBE(&byte_buffer[index * sizeof(float)]);;
            }
            index++;
            if(subband < MAX_SUBBANDS){
              if (sample < 0.0f){
                initial_flagged[second][subband][channel] = TRUE;
                flagged[second][subband][channel] = TRUE;
              } else {
                data[second][subband][pol][channel] += sample;
              }
            }
          }
        }
      }
    }
  }

  data_info = (Data_info*) malloc(sizeof(Data_info));
  data_info->nr_stations         = nr_stations;
  data_info->nr_times            = nr_times;
  data_info->nr_subbands         = nr_subbands;
  data_info->nr_subbands_in_file = nr_subbands_in_file;
  data_info->nr_channels         = nr_channels;
  data_info->integration_factor  = integration_factor;
  data_info->nr_polarizations    = nr_polarizations;
  data_info->station_block_size  = station_block_size;
  data_info->data                = data;
  data_info->flagged             = flagged;
  data_info->initial_flagged     = initial_flagged;


  remove_initial_flagged(data_info);

  return data_info;
}

Data_info* fake_data(int nr_times, int nr_subbands, int nr_channels, int nr_polarizations){
  Data_info* result = (Data_info*) malloc(sizeof(Data_info));
  float**** data;
  unsigned char*** flagged;
  unsigned char*** initial_flagged;
  int t, subband, channel, pol;
  float sample;

  srand((unsigned int)time(NULL));
  
  allocate_memory(&data, &flagged, &initial_flagged, nr_times, nr_subbands, nr_channels, nr_polarizations);
  for(t = 0; t < nr_times; t++){
    for(subband = 0; subband < nr_subbands; subband++){
      for(channel = 0; channel < nr_channels; channel++){
        for(pol = 0; pol < nr_polarizations; pol++){
          sample = (float)rand()/(float)(RAND_MAX/2493000000.0f) + 7000000.0f;
          data[t][subband][pol][channel] += sample;
        }
      }
    }
  }

  result->nr_times            = nr_times;
  result->nr_subbands         = nr_subbands;
  result->nr_channels         = nr_channels;
  result->nr_polarizations    = nr_polarizations;
  result->data                = data;
  result->flagged             = flagged;
  result->initial_flagged     = initial_flagged;

  return result;
}
  

Device_array_pointers* allocate_cuda_memory(Data_info* h_data_info){
  int nr_times = h_data_info->nr_times;
  int nr_subbands = h_data_info->nr_subbands;
  int nr_channels = h_data_info->nr_channels;
  int nr_polarizations = h_data_info->nr_polarizations;
  int i,j,k,l;
  int index;
  hipError_t cErr;
  unsigned char* flagged;
  unsigned char* initial_flagged;
  Device_array_pointers* result;


  result = (Device_array_pointers*) malloc(sizeof(Device_array_pointers));
  if(result == NULL){
    perror("Error allocating memory for device array pointers");
    exit(1);
  }

  /* Allocate memory on host */
  float* linear_data = (float *)malloc(sizeof(float) * (nr_times * nr_subbands *
        nr_channels * nr_polarizations));
  if(linear_data == NULL){
    perror("Error allocating memory for linear_data");
    exit(1);
  }

  unsigned char* linear_flagged = (unsigned char*)malloc(sizeof(unsigned char) * (nr_times *
        nr_subbands * nr_channels * nr_polarizations));
  if(linear_flagged == NULL){
    perror("Error allocating memory for linear_flagged");
    exit(1);
  }

  unsigned char* linear_initial_flagged = (unsigned char*)malloc(sizeof(unsigned char) * (nr_times *
        nr_subbands * nr_channels * nr_polarizations));
  if(linear_initial_flagged == NULL){
    perror("Error allocating memory for linear_initial_flagged");
    exit(1);
  }

  /* Allocate memory on device */
  if( (cErr = hipMalloc(&result->data, sizeof(float) * (nr_times * nr_subbands *
        nr_channels * nr_polarizations))) != hipSuccess){
   fprintf(stderr, "Error allocating device memory for data.\n");
   fprintf(stderr, "%s\n", hipGetErrorString(cErr));
   exit(1);
  }

  if( (cErr = hipMalloc(&flagged, sizeof(unsigned char) * (nr_times * nr_subbands *
        nr_channels * nr_polarizations))) != hipSuccess){
   fprintf(stderr, "Error allocating device memory for flags.\n");
   fprintf(stderr, "%s\n", hipGetErrorString(cErr));
   exit(1);
  }

  if( (cErr = hipMalloc(&initial_flagged, sizeof(unsigned char) * (nr_times * nr_subbands *
        nr_channels * nr_polarizations))) != hipSuccess){
   fprintf(stderr, "Error allocating device memory for initial flags.\n");
   fprintf(stderr, "%s\n", hipGetErrorString(cErr));
   exit(1);
  }

  //printf("data size = %d\n", nr_times * nr_subbands * nr_channels *
      //nr_polarizations);


  /* Fill linearized array */
  for(i=0;i<nr_times;i++){
    for(j=0;j<nr_subbands;j++){
      for(k=0;k<nr_polarizations;k++){
        for(l=0;l<nr_channels;l++){
          index = (i * nr_subbands * nr_polarizations * nr_channels) 
            + (j * nr_polarizations * nr_channels) 
            + (k * nr_channels) + l;
          linear_data[index] = h_data_info->data[i][j][k][l];
          linear_flagged[index] = h_data_info->flagged[i][j][k];
          linear_initial_flagged[index] = h_data_info->initial_flagged[i][j][k];
        }
      }
    }
  }


  /* Copy linearized arrays to device */

  if( (cErr = hipMemcpy(result->data, linear_data, sizeof(float) * (nr_times * nr_subbands *
        nr_channels * nr_polarizations), hipMemcpyHostToDevice)) !=
        hipSuccess){
    fprintf(stderr, "Error copying data to device:\n");
    fprintf(stderr, "%s\n", hipGetErrorString(cErr));
    exit(1);
  }

  if( (cErr = hipMemcpy(flagged, linear_flagged, sizeof(unsigned char) * (nr_times * nr_subbands *
        nr_channels * nr_polarizations ), hipMemcpyHostToDevice)) !=
        hipSuccess){
    fprintf(stderr, "Error copying flags to device:\n");
    fprintf(stderr, "%s\n", hipGetErrorString(cErr));
    exit(1);
  }

  if( (cErr = hipMemcpy(initial_flagged, linear_initial_flagged, sizeof(unsigned char) * (nr_times * nr_subbands *
        nr_channels * nr_polarizations ), hipMemcpyHostToDevice)) !=
        hipSuccess){
    fprintf(stderr, "Error copying flags to device:\n");
    fprintf(stderr, "%s\n", hipGetErrorString(cErr));
    exit(1);
  }



  result->flagged = flagged;
  result->initial_flagged = initial_flagged;

  return result;
}

Device_array_pointers* malloc_cuda_memory(Data_info* h_data_info, Device_data** d_data){
  hipError_t cErr;

  Device_data* h_data = (Device_data*) malloc(sizeof(Device_data)); 

  h_data->nr_stations         = h_data_info->nr_stations;
  h_data->nr_times            = h_data_info->nr_times;
  h_data->nr_subbands         = h_data_info->nr_subbands;
  h_data->nr_subbands_in_file = h_data_info->nr_subbands_in_file;
  h_data->nr_channels         = h_data_info->nr_channels;
  h_data->integration_factor  = h_data_info->integration_factor;
  h_data->nr_polarizations    = h_data_info->nr_polarizations;
  h_data->station_block_size  = h_data_info->station_block_size;

  if((cErr = hipMalloc(&(*d_data), sizeof(Device_data))) != hipSuccess){
    fprintf(stderr, "Error allocating device memory for data_info struct");
    fprintf(stderr, "%s\n", hipGetErrorString(cErr));
    exit(1);
  }
  if((cErr = hipMemcpy(*d_data, h_data, sizeof(Device_data),
          hipMemcpyHostToDevice)) != hipSuccess){
    fprintf(stderr, "Error copying data_info to device\n");
    fprintf(stderr, "%s\n", hipGetErrorString(cErr));
    exit(1);
  }
  return allocate_cuda_memory(h_data_info);
}


