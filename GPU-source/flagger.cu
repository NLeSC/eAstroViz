#include "hip/hip_runtime.h"
#include <assert.h>
#include <assert.h>
#include <stdio.h>
#include <limits.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <string.h>
#include <stdlib.h>
#include <arpa/inet.h>
#include <hip/hip_runtime.h>
#include "cuPrintf.cu"

#include "Data_reader.h"
#include "Device_data.h"
#include "Data_info.h"
#include "Device_array_pointers.h"


#define MAX_SUBBANDS INT_MAX
#define MAX_SEQ_NR   INT_MAX
#define MAX_THREADS  1024
#define NO_THREADS_FREQ   128
#define NO_THREADS_TIME   256
#define DOWNSAMPLE   4
#define TRUE         1
#define FALSE        0
#define MAX_ITERS    7
#define FIRST_THRESHOLD 6.0f
#define BASE_SENSITIVITY 1.0f
#define SIR_VALUE 0.4f


extern "C" {
  void start_timer();
  void stop_timer(float *time);
}


__device__ inline void swap(float & a, float & b){
  float tmp = a;
  a = b;
  b = tmp;
}

__device__ inline void swap(double & a, double & b){
  float tmp = a;
  a = b;
  b = tmp;
}

__device__ float  bitonicSort(float* values, int n, int nr_flagged){
  const int tid = threadIdx.x;


  //Parallel bitonic sort
  for(int k = 2; k <= n; k *= 2){
    //Bitonic merge;
    for(int j = k/2; j>0;j /= 2){
      int ixj = tid ^ j;
      if (ixj > tid){
        if((tid & k) == 0){
          if(values[tid] > values[ixj]){
            swap(values[tid], values[ixj]);
          }
        }else{
          if(values[tid] < values[ixj]){
            swap(values[tid], values[ixj]);
          }
        }
      }
      __syncthreads();
    }
  }
  return values[nr_flagged + (n - nr_flagged)/2];
}

__device__ double bitonicSort(double* values, int n, int nr_flagged){
  const int tid = threadIdx.x;


  //Parallel bitonic sort
  for(int k = 2; k <= n; k *= 2){
    //Bitonic merge;
    for(int j = k/2; j>0;j /= 2){
      int ixj = tid ^ j;
      if (ixj > tid){
        if((tid & k) == 0){
          if(values[tid] > values[ixj]){
            swap(values[tid], values[ixj]);
          }
        }else{
          if(values[tid] < values[ixj]){
            swap(values[tid], values[ixj]);
          }
        }
      }
      __syncthreads();
    }
  }
  return (double)values[nr_flagged + (n - nr_flagged)/2];
}

__device__ float sum_values(float* values){

  unsigned int tid = threadIdx.x;

  for(unsigned int s=blockDim.x/2; s > 32; s>>=1){
    if(tid < s){
      values[tid] += values[tid + s];
    }
    __syncthreads();
  }

  if(tid < 32){
    values[tid] += values[tid + 32];
    values[tid] += values[tid + 16];
    values[tid] += values[tid + 8];
    values[tid] += values[tid + 4];
    values[tid] += values[tid + 2];
    values[tid] += values[tid + 1];
  }

  return values[0];
}

__device__ double sum_values(double* values){

  unsigned int tid = threadIdx.x;

  for(unsigned int s=blockDim.x/2; s > 0; s>>=1){
    if(tid < s){
      values[tid] += values[tid + s];
    }
    __syncthreads();
  }

  return values[0];
}

__device__ void count_flags(unsigned int* nr_flagged, unsigned char* flags){
  unsigned int tid = threadIdx.x;
  if(flags[tid] == TRUE){
    atomicAdd(nr_flagged, 1);
  }
}


__device__ void sum_threshold(float* values, unsigned char* flags, float median, float stddev, int n){
  int window = 1;
  int tid = threadIdx.x;
  float factor = stddev * BASE_SENSITIVITY;
  float sum;
  int pos;
  float threshold;


  for(int i=0;i<MAX_ITERS;i++){
    threshold = fma((FIRST_THRESHOLD * powf(1.5f, i)/ window),factor, median);
    sum = 0.0f;
    if(tid % window == 0){
      for(pos = tid; pos < tid + window; pos++){
        if(flags[pos] != TRUE){
          sum += values[pos];
        }else{
          sum += threshold;
        }
      }
      if(sum >= window * threshold)
        for(pos = tid; pos < tid + window; pos++){
          flags[pos] = TRUE;
        }
      }
    window *= 2;
  }
}

__device__ void sum_threshold(double* values, unsigned char* flags, float median, float stddev, int n){
  int window = 1;
  int tid = threadIdx.x;
  float factor = stddev * BASE_SENSITIVITY;
  float sum;
  int pos;
  float threshold;


  for(int i=0;i<MAX_ITERS;i++){
    threshold = fma((FIRST_THRESHOLD * powf(1.5f, i)/ window),factor, median);
    sum = 0.0f;
    if(tid % window == 0){
      for(pos = tid; pos < tid + window; pos++){
        if(flags[pos] != TRUE){
          sum += values[pos];
        }else{
          sum += threshold;
        }
      }
      if(sum >= window * threshold)
        for(pos = tid; pos < tid + window; pos++){
          flags[pos] = TRUE;
        }
      }
    window *= 2;
  }
}
	
    
        
__global__ void sir_operator(unsigned char* d_flags, int n){
  unsigned char* flags = &(d_flags[(blockIdx.x * n)]);
  float credit = 0.0f;
  float w;
  float max_credit0;
  for(int i = 0; i < n;i++){
    w = flags[i] ? SIR_VALUE : SIR_VALUE - 1.0f;
    max_credit0 = credit > 0.0f ? credit : 0.0f;
    credit = max_credit0 + w;
    flags[i] = credit >= 0.0f;
  }
  credit = 0;
  for(int i = n-1; i > 0;i--){
    w = flags[i] ? SIR_VALUE : SIR_VALUE - 1.0f;
    max_credit0 = credit > 0.0f ? credit : 0.0f;
    credit = max_credit0 + w;
    flags[i] = credit >= 0.0f | flags[i];
  }
}
      
  
// This method reduces all channels in all subbands in a data set to single 
// values. It stores this on the device memory so it can later be flagged.
// It groups values belonging to the same polarization together for 
// easier processing.

__global__ void Reduce_freq(float* values, int nr_blocks, float* results, Device_data* d_data){

  extern __shared__ float shared[];
  shared[threadIdx.x] = values[(blockIdx.x * blockDim.x) + threadIdx.x];
  int pol = blockIdx.x % d_data->nr_polarizations;
  int pos = blockIdx.x / d_data->nr_polarizations;
  int offset = pol * d_data->nr_subbands;
  results[pos + offset] = sum_values(shared);
}

  

// Replace top and bottom 10% with value at border. 
// As the flagged values
// are in front and the first nr_flagged threads don't
// participate, they are not taken into concideration for 
// the top and bottom 10 percentiles.

__device__ void winsorize(float* shared, int nr_flagged, int n){
    if(threadIdx.x < (0.1f * (n - nr_flagged) + nr_flagged)){ 
      shared[threadIdx.x] = shared[(int)(0.1f * (n - nr_flagged) + nr_flagged)];
    }
    if(threadIdx.x > (0.9f * (n - nr_flagged) + nr_flagged)){
      shared[threadIdx.x] = shared[(int)(0.9f * (n - nr_flagged) + nr_flagged)];
    }
}

__device__ void winsorize(double* shared, int nr_flagged, int n){
    if(threadIdx.x < (0.1f * (n - nr_flagged) + nr_flagged)){ 
      shared[threadIdx.x] = shared[(int)(0.1f * (n - nr_flagged) + nr_flagged)];
    }
    if(threadIdx.x > (0.9f * (n - nr_flagged) + nr_flagged)){
      shared[threadIdx.x] = shared[(int)(0.9f * (n - nr_flagged) + nr_flagged)];
    }
}

__device__ __inline__ float get_value(int time, int subband, int pol, int channel, float*
                          values, Device_data* d_data ){
  return values[(time * d_data->nr_subbands * d_data->nr_polarizations * d_data->nr_channels) +
                (subband * d_data->nr_polarizations * d_data->nr_channels) + 
                (pol * d_data->nr_channels) + 
                channel];
}

__device__ __inline__ unsigned char get_value(int time, int subband, int pol, int channel,
                                   unsigned char* values, Device_data* d_data ){
  return values[(time * d_data->nr_subbands * d_data->nr_polarizations * d_data->nr_channels) +
                (subband * d_data->nr_polarizations * d_data->nr_channels) + 
                (pol * d_data->nr_channels) + 
                channel];
}

__device__ __inline__ void set_value(unsigned char value, int time, int subband, int pol, int channel, 
                          unsigned char* values, Device_data* d_data ){
  values[(time * d_data->nr_subbands * d_data->nr_polarizations * d_data->nr_channels) +
         (subband * d_data->nr_polarizations * d_data->nr_channels) + 
         (pol * d_data->nr_channels) + 
         channel] = (value | get_value(time, subband, pol, channel,
                                      values, d_data));
}

__global__ void Reduce_time(float* values, int nr_blocks, int nr_threads, float* results, Device_data* d_data){
  extern __shared__ float shared[];

  int channel = blockIdx.x % d_data->nr_channels;
  int pol = blockIdx.x / d_data->nr_channels % d_data->nr_polarizations;
  int subband = blockIdx.x / (d_data->nr_channels * d_data->nr_polarizations) % d_data->nr_subbands;
  int time = (blockIdx.x / (d_data->nr_channels * d_data->nr_polarizations *
            d_data->nr_subbands)) * nr_threads + threadIdx.x;

  shared[threadIdx.x] = get_value(time, subband, pol, channel, values, d_data);
  results[blockIdx.x] = sum_values(shared);
}
  

__global__ void Flagger_freq(float* values, unsigned char* d_flags, unsigned int n,
    unsigned int* d_nr_flagged){
  
  extern __shared__ float shared[];
  unsigned char* flags;
  unsigned int tid = threadIdx.x;
  float median;
  float stddev;
  int i;
  
  // Copy input to shared memory
  shared[tid] = values[(blockIdx.x * blockDim.x) + tid];
  flags = (unsigned char*) &shared[n];
  flags[tid] = (unsigned char)0;

  __syncthreads();

  for(i=0; i < 2; i++){
    int nr_flagged = d_nr_flagged[blockIdx.x];

    // the sort method will move all flagged values,
    // which have been set to zero, to the front of 
    // the array.
    median = bitonicSort(shared, n, nr_flagged);

    if(tid >= nr_flagged){
      winsorize(shared, nr_flagged, n);
    }
    __syncthreads();

    // Calculate the sum of all values
    float sum = sum_values(shared);

    // Reset values as reduce alters them, and set flagged values 
    // to zero.
    shared[tid] = values[(blockIdx.x * blockDim.x) + tid];
    if(flags[tid]){
      shared[tid] = 0.0f;
    }
    __syncthreads();

    // And sort them again
    bitonicSort(shared, n, nr_flagged);
    
    if(tid >= nr_flagged){
      winsorize(shared, nr_flagged, n);
      // Square the values
      shared[tid] *= shared[tid];
    }
    __syncthreads();

    // Calculate the sum of squares
    float squaredSum = sum_values(shared);

    stddev = sqrtf(squaredSum/n - (sum/n * sum/n));

    // Reset values
    shared[tid] = values[(blockIdx.x * blockDim.x) + tid];
    if(flags[tid]){
      shared[tid] = 0.0f;
    }
    __syncthreads();

    sum_threshold(shared, flags, median, stddev, n); 

    // Reset and recount the number of flags
    d_nr_flagged[blockIdx.x] = 0;
    count_flags(&(d_nr_flagged[blockIdx.x]), flags);
  }

  d_flags[(blockIdx.x * blockDim.x) + tid] = (d_flags[(blockIdx.x *
    blockDim.x)+ tid] | flags[tid]);
}

__global__ void Flagger_time(float* values, unsigned char* d_flags, unsigned int n,
    unsigned int m, unsigned int* d_nr_flagged, Device_data* d_data){

  extern __shared__ float shared[];
  unsigned char* flags;
  unsigned int tid = threadIdx.x;
  float median;
  float stddev;
  int i;
  
  // Copy input to shared memory
  // M is the number of blocks. This is not coalesced
  // memory access, but the data structure leaves us 
  // no other choice.

  int channel = blockIdx.x % d_data->nr_channels;
  int pol = blockIdx.x / d_data->nr_channels % d_data->nr_polarizations;
  int subband = blockIdx.x / (d_data->nr_channels * d_data->nr_polarizations) % d_data->nr_subbands;
  int time = (blockIdx.x / (d_data->nr_channels * d_data->nr_polarizations *
            d_data->nr_subbands)) * n + tid;

  shared[tid] = get_value(time, subband, pol, channel, values, d_data);

  flags = (unsigned char*) &shared[n];
  flags[tid] = get_value(time, subband, pol, channel, d_flags, d_data);

  d_nr_flagged[blockIdx.x] = 0;
  count_flags(&(d_nr_flagged[blockIdx.x]), flags);

  __syncthreads();

  for(i=0; i < 2; i++){
    int nr_flagged = d_nr_flagged[blockIdx.x];

    // the sort method will move all flagged values,
    // which have been set to zero, to the front of 
    // the array.
    median = bitonicSort(shared, n, nr_flagged);

    if(tid >= nr_flagged){
      winsorize(shared, nr_flagged, n);  
    }
    __syncthreads();

    float sum = sum_values(shared);

    // Reset values as reduce alters them, and set flagged values 
    // to zero.
    shared[tid] = get_value(time, subband, pol, channel, values, d_data);
    if(flags[tid]){
      shared[tid] = 0.0f;
    }
    __syncthreads();
    // And sort them again
    bitonicSort(shared, n, nr_flagged);
    
    if(tid >= nr_flagged){
      winsorize(shared, nr_flagged, n);  
      // Square the values
      shared[tid] *= shared[tid];
    }
    __syncthreads();

    float squaredSum = sum_values(shared);

    stddev = sqrtf(squaredSum/n - (sum/n * sum/n));

    // Reset values
    shared[tid] = get_value(time, subband, pol, channel, values, d_data);
    if(flags[tid]){
      shared[tid] = 0.0f;
    }
    __syncthreads();

    sum_threshold(shared, flags, median, stddev, n); 

    // Reset and recount the number of flags
    d_nr_flagged[blockIdx.x] = 0;
    count_flags(&(d_nr_flagged[blockIdx.x]), flags);
  }


  set_value(flags[tid], time, subband, pol, channel, d_flags, d_data);
}

__global__ void Flagger_time_reduced(float* values, unsigned char* d_flags, unsigned int n,
    unsigned int m, unsigned int* d_nr_flagged, Device_data* d_data){
  extern __shared__ double d_shared[];
  unsigned char* flags;
  unsigned int tid = threadIdx.x;
  float median;
  float stddev;
  int i;
  
  // Copy input to shared memory
  int channel = blockIdx.x % d_data->nr_channels;
  int pol = blockIdx.x / d_data->nr_channels % d_data->nr_polarizations;
  int subband = blockIdx.x / (d_data->nr_channels * d_data->nr_polarizations) % d_data->nr_subbands;
  int time = (blockIdx.x / (d_data->nr_channels * d_data->nr_polarizations *
            d_data->nr_subbands)) * n + tid;

  d_shared[tid] = (double)get_value(time, subband, pol, channel, values, d_data);
  flags = (unsigned char*) &d_shared[n];
  flags[tid] = (unsigned char)0;
  /*
  flags[tid] = get_value(time, subband, pol, channel, d_flags, d_data);

  d_nr_flagged[blockIdx.x] = 0;
  count_flags(&(d_nr_flagged[blockIdx.x]), flags);

  if(tid == 0){
    cuPrintf("block: %d, nr_flagged: %d\n", blockIdx.x,
    d_nr_flagged[blockIdx.x]);
  }
  */

  __syncthreads();

  for(i=0; i < 2; i++){
    int nr_flagged = d_nr_flagged[blockIdx.x];

    // the sort method will move all flagged values,
    // which have been set to zero, to the front of 
    // the array.
    median = bitonicSort(d_shared, n, nr_flagged);

    if(tid >= nr_flagged){
      winsorize(d_shared, nr_flagged, n);
    }
    __syncthreads();

    double sum = sum_values(d_shared);

    // Reset values as reduce alters them, and set flagged values 
    // to zero.
    d_shared[tid] = (double)get_value(time, subband, pol, channel, values, d_data);
    if(flags[tid]){
      d_shared[tid] = 0.0f;
    }
    __syncthreads();
    // And sort them again
    bitonicSort(d_shared, n, nr_flagged);
    
    if(tid >= nr_flagged){
      winsorize(d_shared, nr_flagged, n);
      // Square the values
      d_shared[tid] = d_shared[tid] * d_shared[tid];
    }

    __syncthreads();
    double squaredSum = sum_values(d_shared);

    stddev = sqrtf(squaredSum/n - (sum/n * sum/n));

    // Reset values
    d_shared[tid] = (double)get_value(time, subband, pol, channel, values, d_data);
    if(flags[tid]){
      d_shared[tid] = 0.0f;
    }
    __syncthreads();

    sum_threshold(d_shared, flags, median, stddev, n); 

    // Reset and recount the number of flags
    d_nr_flagged[blockIdx.x] = 0;
    count_flags(&(d_nr_flagged[blockIdx.x]), flags);
  }

  if(flags[tid]){
    for(i=0;i<DOWNSAMPLE;i++){
      set_value((unsigned char)1, (time * DOWNSAMPLE) + i , subband, pol,
      channel, d_flags, d_data);
    }
  }
}
__global__ void Flagger_freq_reduced(float* values, unsigned char* d_flags, unsigned int n,
    unsigned int m, unsigned int* d_nr_flagged, Device_data* d_data){
  extern __shared__ double d_shared[];
  unsigned char* flags;
  unsigned int tid = threadIdx.x;
  float median;
  float stddev;
  int i;
  
  // Copy input to shared memory
  d_shared[tid] = (double)values[(blockIdx.x * blockDim.x) + tid];
  flags = (unsigned char*) &d_shared[n];
  flags[tid] = (unsigned char)0;


  __syncthreads();

  for(i=0; i < 2; i++){
    int nr_flagged = d_nr_flagged[blockIdx.x];

    // the sort method will move all flagged values,
    // which have been set to zero, to the front of 
    // the array.
    median = bitonicSort(d_shared, n, nr_flagged);

    if(tid >= nr_flagged){
      winsorize(d_shared, nr_flagged, n);
    }
    __syncthreads();

    double sum = sum_values(d_shared);

    // Reset values as reduce alters them, and set flagged values 
    // to zero.
    d_shared[tid] = values[(blockIdx.x * blockDim.x) + tid];
    if(flags[tid]){
      d_shared[tid] = 0.0f;
    }
    __syncthreads();
    // And sort them again
    bitonicSort(d_shared, n, nr_flagged);
    
    if(tid >= nr_flagged){
      winsorize(d_shared, nr_flagged, n);
      // Square the values
      d_shared[tid] = d_shared[tid] * d_shared[tid];
    }

    __syncthreads();
    double squaredSum = sum_values(d_shared);

    stddev = sqrtf(squaredSum/n - (sum/n * sum/n));

    // Reset values
    d_shared[tid] = values[(blockIdx.x * blockDim.x) + tid];
    if(flags[tid]){
      d_shared[tid] = 0.0f;
    }
    __syncthreads();

    sum_threshold(d_shared, flags, median, stddev, n); 

    // Reset and recount the number of flags
    d_nr_flagged[blockIdx.x] = 0;
    count_flags(&(d_nr_flagged[blockIdx.x]), flags);
  }

  int pol = blockIdx.x % d_data->nr_polarizations;
  int timeslot = (blockIdx.x/d_data->nr_polarizations) * (d_data->nr_subbands * d_data->nr_polarizations * d_data->nr_channels);
  int subband = (threadIdx.x * d_data->nr_polarizations * d_data->nr_channels) + (pol * d_data->nr_channels);

  if(flags[tid]){
    for(i=0;i<d_data->nr_channels;i++){
      d_flags[timeslot + subband + i] = (unsigned char)1;
    }
  }
}

void merge_flagmap(unsigned char* flagmap, int n, int pols, int nr_channels){
  int i,j,k;
  // You cannot merge a flagmap with only one polarisation
  if(pols == 1){
    return;
  }

  for(i=0;i<n/pols;i++){
    for(j=0;j<pols;j++){
      // The next polarisation is always `nr_channels` values away
      if(flagmap[n + j * nr_channels]){
    	for(k=0;k<pols;k++){
	  flagmap[n + k * nr_channels] = (unsigned char) 1;
        }
      }
    }
  }
}
   

void get_and_check_arguments(int argc, char** argv, int* direction, int* reduce, int* little_endian, int* sir){
  int i;

  if(argc > 6 || argc < 2){
    printf("Usage: flagger <filename> [--little-endian] [-f | -t | -b] [-r] [-sir]\n");
    exit(1);
  }

  if(argc > 2){
    for(i=2;i<argc;i++){
      if (strcmp(argv[i], "-f") == 0){
        *direction = 0;
        continue;
      }
      if (strcmp(argv[i], "-t") == 0){
        *direction = 1;
        continue;
      }
      if (strcmp(argv[i], "-b") == 0){
        *direction = 2;
        continue;
      }
      if (strcmp(argv[i], "-b2") == 0){
        *direction = 3;
        continue;
      }
      if (strcmp(argv[i], "-r") == 0){
        *reduce = 1;
        continue;
      }
      if (strcmp(argv[i], "--little-endian") == 0){
        *little_endian = 1;
        continue;
      }
      if (strcmp(argv[i], "-sir") == 0){
        *sir = 1;
        continue;
      }else{
        fprintf(stderr, "unrecognized option %s.\n", argv[i]);
        exit(1);
      }
    }
  }
}

void allocate_and_zero_memory(unsigned char** d_flags, unsigned int** d_nr_flagged, int data_size, int nr_blocks){
  hipError_t devRetVal;

  devRetVal = hipMalloc(&(*d_flags), (size_t)(data_size *  sizeof(unsigned char)));
  if(devRetVal != hipSuccess){
    fprintf(stderr, "Error allocating memory for flags\n");
    fprintf(stderr, "%s\n", hipGetErrorString(devRetVal));
    exit(1);
  }

  devRetVal = hipMalloc(&(*d_nr_flagged), (size_t)(nr_blocks *  sizeof(unsigned int)));
  if(devRetVal != hipSuccess){
    fprintf(stderr, "Error allocating memory for n's\n");
    fprintf(stderr, "%s\n", hipGetErrorString(devRetVal));
    exit(1);
  }

  devRetVal = hipMemset(*d_nr_flagged, 0, (size_t)(nr_blocks *  sizeof(unsigned int)));
  if(devRetVal != hipSuccess){
    fprintf(stderr, "Error setting initial n values\n");
    fprintf(stderr, "%s\n", hipGetErrorString(devRetVal));
    exit(1);
  }

  devRetVal = hipMemset(*d_flags, 0, (size_t)(data_size * sizeof(unsigned
  char)));
  if(devRetVal != hipSuccess){
    fprintf(stderr, "Error zero'ing flagmap\n");
    fprintf(stderr, "%s\n", hipGetErrorString(devRetVal));
    exit(1);
  }
}

void allocate_nr_flagged(unsigned int** d_nr_flagged, int nr_blocks){
  hipError_t devRetVal;


  devRetVal = hipMalloc(&(*d_nr_flagged), (size_t)(nr_blocks *  sizeof(unsigned int)));
  if(devRetVal != hipSuccess){
    fprintf(stderr, "Error allocating memory for n's\n");
    fprintf(stderr, "%s\n", hipGetErrorString(devRetVal));
    exit(1);
  }
}


unsigned char* frequency_flagger(int data_size, unsigned char* d_flags, Data_info* h_data_info, Device_array_pointers* ptrs){
  int nr_blocks;
  int shared_mem_size; 
  int nr_threads;
  hipError_t devRetVal;
  unsigned int *d_nr_flagged = NULL;

  nr_threads = h_data_info->nr_channels;
  shared_mem_size = nr_threads * sizeof(float) + 
                    nr_threads * sizeof(unsigned char);
  nr_blocks = h_data_info->nr_times * h_data_info->nr_subbands *
              h_data_info->nr_polarizations;

  allocate_and_zero_memory(&d_flags, &d_nr_flagged, data_size, nr_blocks);

  Flagger_freq<<<nr_blocks, nr_threads, shared_mem_size>>>
    (ptrs->data, d_flags, nr_threads, d_nr_flagged);

  if( (devRetVal = hipGetLastError()) != hipSuccess){
    fprintf(stderr, "Kernel has some kind of issue:\n%s\n",
        hipGetErrorString(devRetVal));
    exit(1);
  }

  return d_flags;
}

unsigned char* time_flagger(int data_size, unsigned char* d_flags, 
                            Data_info* h_data_info, Device_array_pointers* ptrs,
                            Device_data* d_data, int chain){
  int nr_blocks;
  int shared_mem_size; 
  int nr_threads;
  hipError_t devRetVal;
  unsigned int *d_nr_flagged = NULL;

  nr_threads = NO_THREADS_TIME;
  shared_mem_size = nr_threads * sizeof(float) + 
                    nr_threads * sizeof(unsigned char);
  nr_blocks = h_data_info->nr_channels * h_data_info->nr_polarizations *
              h_data_info->nr_subbands * (h_data_info->nr_times / nr_threads);

  if(!chain){
    allocate_and_zero_memory(&d_flags, &d_nr_flagged, data_size, nr_blocks);
  }else{
    allocate_nr_flagged(&d_nr_flagged, nr_blocks);
  }

  Flagger_time<<<nr_blocks, nr_threads, shared_mem_size>>>
    (ptrs->data, d_flags, nr_threads, nr_blocks, d_nr_flagged, d_data);

  if( (devRetVal = hipGetLastError()) != hipSuccess){
    fprintf(stderr, "Kernel has some kind of issue:\n%s\n",
        hipGetErrorString(devRetVal));
    exit(1);
  }

  return d_flags;
}

float* reduce_frequency(int data_size, Data_info* h_data_info, Device_array_pointers* ptrs, Device_data* d_data){
  hipError_t devRetVal;
  int nr_blocks;
  int nr_threads;
  int shared_mem_size;
  float* d_values_reduced;

  nr_threads = h_data_info->nr_channels;
  nr_blocks = data_size/nr_threads;
  shared_mem_size = nr_threads * sizeof(float);

  devRetVal = hipMalloc(&d_values_reduced, (size_t)((data_size /
  h_data_info->nr_channels) * sizeof(float)));
  if(devRetVal != hipSuccess){
    fprintf(stderr, "Error allocating memory for reduced values\n");
    fprintf(stderr, "%s\n", hipGetErrorString(devRetVal));
    exit(1);
  }

  Reduce_freq<<<nr_blocks, nr_threads, shared_mem_size>>>
    (ptrs->data, data_size/h_data_info->nr_channels, d_values_reduced, d_data);

  if( (devRetVal = hipGetLastError()) != hipSuccess){
    fprintf(stderr, "Kernel has some kind of issue:\n%s\n",
        hipGetErrorString(devRetVal));
    exit(1);
  }

  return d_values_reduced;
}

float* reduce_time(int data_size, Data_info* h_data_info, Device_array_pointers* ptrs, Device_data* d_data){
  hipError_t devRetVal;
  int nr_blocks;
  int nr_threads;
  int shared_mem_size;
  float* d_values_reduced;

  nr_threads = DOWNSAMPLE;
  nr_blocks = h_data_info->nr_channels * h_data_info->nr_polarizations *
              h_data_info->nr_subbands * (h_data_info->nr_times / nr_threads);
  shared_mem_size = nr_threads * sizeof(float);

  devRetVal = hipMalloc(&d_values_reduced, (size_t)((data_size /
  DOWNSAMPLE) * sizeof(float)));
  if(devRetVal != hipSuccess){
    fprintf(stderr, "Error allocating memory for reduced values\n");
    fprintf(stderr, "%s\n", hipGetErrorString(devRetVal));
    exit(1);
  }

  Reduce_time<<<nr_blocks, nr_threads, shared_mem_size>>>
    (ptrs->data, nr_blocks, nr_threads, d_values_reduced, d_data);

  if( (devRetVal = hipGetLastError()) != hipSuccess){
    fprintf(stderr, "Kernel has some kind of issue:\n%s\n",
        hipGetErrorString(devRetVal));
    exit(1);
  }

  return d_values_reduced;
}


unsigned char* frequency_flagger_reduced(int data_size, unsigned char* d_flags, 
                               Data_info* h_data_info, 
                               Device_array_pointers* ptrs, 
                               Device_data* d_data){
  int nr_blocks;
  int shared_mem_size; 
  int nr_threads;
  hipError_t devRetVal;
  unsigned int *d_nr_flagged = NULL;
  float* d_values_reduced = NULL;

  d_values_reduced = reduce_frequency(data_size, h_data_info, ptrs, d_data);

  nr_threads = h_data_info->nr_subbands;
  nr_blocks = h_data_info->nr_times * h_data_info->nr_polarizations;
  shared_mem_size = sizeof(double) * nr_threads + 
                    sizeof(unsigned char) * nr_threads;

  allocate_and_zero_memory(&d_flags, &d_nr_flagged, data_size, nr_blocks);

  Flagger_freq_reduced<<<nr_blocks, nr_threads, shared_mem_size>>>
    (d_values_reduced, d_flags, nr_threads, nr_blocks, d_nr_flagged, d_data);

  if( (devRetVal = hipGetLastError()) != hipSuccess){
    fprintf(stderr, "Kernel has some kind of issue:\n%s\n",
        hipGetErrorString(devRetVal));
    exit(1);
  }

  return d_flags;
}

unsigned char* time_flagger_reduced(int data_size, unsigned char* d_flags, 
                               Data_info* h_data_info, 
                               Device_array_pointers* ptrs, 
                               Device_data* d_data,
                               int chain){
  int nr_blocks;
  int shared_mem_size; 
  int nr_threads;
  hipError_t devRetVal;
  unsigned int *d_nr_flagged = NULL;
  float* d_values_reduced = NULL;

  d_values_reduced = reduce_time(data_size, h_data_info, ptrs, d_data);

  nr_threads = NO_THREADS_FREQ;
  nr_blocks = h_data_info->nr_channels * h_data_info->nr_polarizations *
  h_data_info->nr_subbands * (h_data_info->nr_times / DOWNSAMPLE / nr_threads);
  shared_mem_size = sizeof(double) * nr_threads + 
                    sizeof(unsigned char) * nr_threads;

  if(!chain){
    allocate_and_zero_memory(&d_flags, &d_nr_flagged, data_size, nr_blocks);
  }else{
    allocate_nr_flagged(&d_nr_flagged, nr_blocks);
  }


  Flagger_time_reduced<<<nr_blocks, nr_threads, shared_mem_size>>>
    (d_values_reduced, d_flags, nr_threads, nr_blocks, d_nr_flagged, d_data);

  if( (devRetVal = hipGetLastError()) != hipSuccess){
    fprintf(stderr, "Kernel has some kind of issue:\n%s\n",
        hipGetErrorString(devRetVal));
    exit(1);
  }

  return d_flags;
}


int main(int argc, char** argv){
  Data_info* h_data_info = NULL;
  Device_data* d_data = NULL;
  Device_array_pointers* ptrs = NULL;
  float time;
  int i,j;
  int nr_blocks;
  unsigned int *d_nr_flagged;
  int data_size;
  unsigned char* d_flags = NULL;
  unsigned char* h_flags;
  float * h_values;
  hipError_t devRetVal;
  int* meta_data = NULL;
  int little_endian = 0;
  int direction = 0;
  int reduce = 0;
  int sir = 0;

  get_and_check_arguments(argc, argv, &direction, &reduce, &little_endian, &sir);


  /*
  int a = 50 * 1024;
  int b = 32;
  int c = 256;
  int d = 1;
  h_data_info = fake_data(a,b,c,d);
  */

  //printf("analyzing %d samples\n", a * b * c *d);

  // Raw files are stored in little endian, and do not contain
  // the necessary meta-data. That's why it's hard-coded here
  if(little_endian){
    meta_data = (int*)malloc(5 * sizeof(int));
    meta_data[0] = 1;   //stations
    meta_data[1] = 59 * 762;  //seconds
    meta_data[2] = 32;  //subbands
    meta_data[3] = 256; //channels
    meta_data[4] = 1;   //polarizations
  }


  h_data_info = read_file(argv[1], meta_data, little_endian);

  ptrs = malloc_cuda_memory(h_data_info, &d_data);

  data_size = h_data_info->nr_times * h_data_info->nr_subbands *
              h_data_info->nr_polarizations * h_data_info->nr_channels;


  start_timer();
  cudaPrintfInit();

  switch(direction){
    case 0:
      if(reduce){
        d_flags = frequency_flagger_reduced(data_size, d_flags, h_data_info, ptrs, d_data); 
      }else{
        d_flags = frequency_flagger(data_size, d_flags, h_data_info, ptrs);
      }
      break;
    case 1:
      if(reduce){
        d_flags = time_flagger_reduced(data_size, d_flags, h_data_info, ptrs,
                                       d_data, FALSE); 
      }else{
        d_flags = time_flagger(data_size, d_flags, h_data_info, ptrs, d_data,
                               FALSE);
      }
      break;
    case 2:
      if(reduce){
        d_flags = frequency_flagger_reduced(data_size, d_flags, h_data_info, ptrs, d_data); 
        d_flags = time_flagger_reduced(data_size, d_flags, h_data_info, ptrs,
                                       d_data, TRUE); 
      }else{
        d_flags = frequency_flagger(data_size, d_flags, h_data_info, ptrs);
        d_flags = time_flagger(data_size, d_flags, h_data_info, ptrs, d_data,
                               TRUE);
      }
      break;
    case 3:
      if(reduce){
        d_flags = time_flagger_reduced(data_size, d_flags, h_data_info, ptrs,
                                       d_data, TRUE); 
        d_flags = frequency_flagger_reduced(data_size, d_flags, h_data_info, ptrs, d_data); 
      }else{
        d_flags = time_flagger(data_size, d_flags, h_data_info, ptrs, d_data,
                               TRUE);
        d_flags = frequency_flagger(data_size, d_flags, h_data_info, ptrs);
      }
      break;
  }

  if(sir){
    nr_blocks = h_data_info->nr_times * h_data_info->nr_subbands * h_data_info->nr_polarizations;
    sir_operator<<<nr_blocks,1>>>(d_flags, h_data_info->nr_channels);
  }


  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();
  hipDeviceSynchronize();

  stop_timer(&time);
  //printf("Kernel ran for %f\t(ms)\n", time);

  if( (devRetVal = hipGetLastError()) != hipSuccess){
    fprintf(stderr, "Kernel has some kind of issue:\n%s\n",
        hipGetErrorString(devRetVal));
    exit(1);
  }
  //printf("calculations done...\n");

  h_flags = (unsigned char *) malloc(sizeof(unsigned char) * data_size);
  if(h_flags == NULL){
    perror("error allocating memory for h_flags:");
    exit(1);
  }

  h_values = (float*) malloc(sizeof(float) * data_size);
  if(h_values == NULL){
    perror("error allocating memory for h_values:");
    exit(1);
  }

  // Copy flags from device back to host
  if( (devRetVal = hipMemcpy(h_flags, d_flags,
          sizeof(unsigned char) * data_size, 
          hipMemcpyDeviceToHost))
        != hipSuccess){
    fprintf(stderr, "Error copying flags from device to host\n");
    fprintf(stderr, "%s\n", hipGetErrorString(devRetVal));
    exit(1);
  }

  // Copy values from device to host
  if( (devRetVal = hipMemcpy(h_values, ptrs->data,
          sizeof(float) * data_size, 
          hipMemcpyDeviceToHost))
        != hipSuccess){
    fprintf(stderr, "Error copying data from device to host\n");
    fprintf(stderr, "%s\n", hipGetErrorString(devRetVal));
    exit(1);
  }


  hipDeviceReset();

  hipFree(d_flags);
  hipFree(d_nr_flagged);



  merge_flagmap(h_flags, data_size, h_data_info->nr_polarizations, h_data_info->nr_channels);
  for(i=0;i<data_size;i++){
    if(h_flags[i]){
	    h_values[i] = 0.0f;
    }
  }

  //printf("%f\n", h_values[0]);
  // Print resulting flagmap
  /*
  int n = h_data_info->nr_channels;
  for(i=0;i< data_size/n ;i += h_data_info->nr_polarizations){
    for(j=0;j<n;j++){
      unsigned char flag = (unsigned char) (h_flags[(i*n) + j] | h_flags[(i*n) + n + j]);
      printf("%u",(unsigned int)flag);
    }
    printf("\n");
  }
  */

  
  fwrite(h_values, data_size, sizeof(float), stdout);


  return 0;
}

